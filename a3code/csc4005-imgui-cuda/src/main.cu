#include "hip/hip_runtime.h"
#include <graphic/graphic.hpp>
#include <imgui_impl_sdl.h>
#include <cstring>
#include <nbody/body.hpp>
#include <chrono>


__global__ void cuda_calaulate(int start_arr[], int idx_arr0[], int idx_arr1[], double m[],double x[],double y[],double vx[],double vy[], double conf[], double x_delta[],double y_delta[],double vx_delta[],double vy_delta[],double ax_delta[],double ay_delta[] ){
    if (blockIdx.x+start_arr[0] < start_arr[1]){
        int i = blockIdx.x+start_arr[0];
        int idx0 = idx_arr0[i];
        int idx1 = idx_arr1[i];
        //std::string str = "thread "+ std::to_string(idx0);

        double delta_x1=0;
        double delta_y1=0;
        double delta_vx1=0;
        double delta_vy1=0;
        double delta_ax1=0;
        double delta_ay1=0;

        double delta_x2=0;
        double delta_y2=0;
        double delta_vx2=0;
        double delta_vy2=0;
        double delta_ax2=0;
        double delta_ay2=0;

        double radius=conf[0];
        double gravity=conf[1];

        double m1=m[idx0];
        double x1=x[idx0];
        double y1=y[idx0];
        double vx1=vx[idx0];
        double vy1=vy[idx0];

        double m2=m[idx1];
        double x2=x[idx1];
        double y2=y[idx1];
        double vx2=vx[idx1];
        double vy2=vy[idx1];

        auto delta_x = x1-x2;
        auto delta_y = y1-y2;
        auto distance_square = delta_x * delta_x + delta_y * delta_y;
        auto ratio = 1 + 0.01;

        if (distance_square < radius * radius) {
            distance_square = radius * radius;
        }
        auto distance = std::sqrt(delta_x * delta_x + delta_y * delta_y);
        if (distance < radius) {
            distance = radius;
        }

        if (delta_x * delta_x + delta_y * delta_y<=radius * radius) {
            auto dot_prod = delta_x * (vx1-vx2)
                            + delta_y * (vy1-vy2);
            auto scalar = 2 / (m1 + m2) * dot_prod / distance_square;

            delta_vx1 = -(scalar * delta_x * m2);
            delta_vy1 = -(scalar * delta_y * m2);
            delta_vx2 = +(scalar * delta_x * m1);
            delta_vy2 = +(scalar * delta_y * m1);
            // now relax the distance a bit: after the collision, there must be
            // at least (ratio * radius) between them
            delta_x1 = +(delta_x / distance * ratio * radius / 2.0);
            delta_y1 = +(delta_y / distance * ratio * radius / 2.0);
            delta_x2 = -(delta_x / distance * ratio * radius / 2.0);
            delta_y2 = -(delta_y / distance * ratio * radius / 2.0);
        }else {
            // update acceleration only when no collision
            auto scalar = gravity / distance_square / distance;
            delta_ax1 = -(scalar * delta_x * m2);
            delta_ay1 = -(scalar * delta_y * m2);
            delta_ax2 = +(scalar * delta_x * m1);
            delta_ay2 = +(scalar * delta_y * m1);
        }

        x_delta[idx0] = delta_x1;
        y_delta[idx0] = delta_y1;
        vx_delta[idx0] = delta_vx1;
        vy_delta[idx0] = delta_vy1;
        ax_delta[idx0] = delta_ax1;
        ay_delta[idx0] = delta_ay1;

        x_delta[idx1] = delta_x2;
        y_delta[idx1] = delta_y2;
        vx_delta[idx1] = delta_vx2;
        vy_delta[idx1] = delta_vy2;
        ax_delta[idx1] = delta_ax2;
        ay_delta[idx1] = delta_ay2;
    }

}

template <typename ...Args>
void UNUSED(Args&&... args [[maybe_unused]]) {}

int main(int argc, char **argv) {
    static int num_threads=4;
    static int bodies = 20;
    if (argc > 3) {
        std::cerr << "wrong arguments. please input only one argument as the number of threads" << std::endl;
        return 0;
    }else if (argc == 3){
        num_threads = std::stoi(argv[1]);
        bodies = std::stoi(argv[2]);
    }


    static float gravity = 100;
    static float space = 800;
    static float radius = 5;
//    static int bodies = 20;
    static float elapse = 0.001;
    static ImVec4 color = ImVec4(1.0f, 1.0f, 0.4f, 1.0f);
    static float max_mass = 50;
    static float current_space = space;
    static float current_max_mass = max_mass;
    static int current_bodies = bodies;
    BodyPool pool(static_cast<size_t>(bodies), space, max_mass);
    graphic::GraphicContext context{"Assignment 2"};
    context.run([&](graphic::GraphicContext *context [[maybe_unused]], SDL_Window *) {
        auto io = ImGui::GetIO();
        ImGui::SetNextWindowPos(ImVec2(0.0f, 0.0f));
        ImGui::SetNextWindowSize(io.DisplaySize);
        ImGui::Begin("Assignment 2", nullptr,
                     ImGuiWindowFlags_NoMove
                     | ImGuiWindowFlags_NoCollapse
                     | ImGuiWindowFlags_NoTitleBar
                     | ImGuiWindowFlags_NoResize);
        ImDrawList *draw_list = ImGui::GetWindowDrawList();
        ImGui::Text("Application average %.3f ms/frame (%.1f FPS)", 1000.0f / ImGui::GetIO().Framerate,
                    ImGui::GetIO().Framerate);
        ImGui::DragFloat("Space", &current_space, 10, 200, 1600, "%f");
        ImGui::DragFloat("Gravity", &gravity, 0.5, 0, 1000, "%f");
        ImGui::DragFloat("Radius", &radius, 0.5, 2, 20, "%f");
        ImGui::DragInt("Bodies", &current_bodies, 1, 2, 100, "%d");
        ImGui::DragFloat("Elapse", &elapse, 0.001, 0.001, 10, "%f");
        ImGui::DragFloat("Max Mass", &current_max_mass, 0.5, 5, 100, "%f");
        ImGui::ColorEdit4("Color", &color.x);
        if (current_space != space || current_bodies != bodies || current_max_mass != max_mass) {
            space = current_space;
            bodies = current_bodies;
            max_mass = current_max_mass;
            pool = BodyPool{static_cast<size_t>(bodies), space, max_mass};
        }


        {
            const ImVec2 p = ImGui::GetCursorScreenPos();
            //pool.update_for_tick(elapse, gravity, space, radius);


            //transform m,x,y,vx,vy,ax,ay
            double *m = (double*)malloc(sizeof(double)*bodies);
            double *x = (double*)malloc(sizeof(double)*bodies);
            double *y = (double*)malloc(sizeof(double)*bodies);
            double *vx = (double*)malloc(sizeof(double)*bodies);
            double *vy = (double*)malloc(sizeof(double)*bodies);
            double *ax = (double*)malloc(sizeof(double)*bodies);
            double *ay = (double*)malloc(sizeof(double)*bodies);

            double *m_delta = (double*)malloc(sizeof(double)*bodies);
            double *x_delta = (double*)malloc(sizeof(double)*bodies);
            double *y_delta = (double*)malloc(sizeof(double)*bodies);
            double *vx_delta = (double*)malloc(sizeof(double)*bodies);
            double *vy_delta = (double*)malloc(sizeof(double)*bodies);
            double *ax_delta = (double*)malloc(sizeof(double)*bodies);
            double *ay_delta = (double*)malloc(sizeof(double)*bodies);

            double *conf = (double*)malloc(sizeof(double)*2);
            for (int i=0; i<bodies;i++){
                m[i]=pool.m[i];
                x[i]=pool.x[i];
                y[i]=pool.y[i];
                vx[i]=pool.vx[i];
                vy[i]=pool.vy[i];
                ax[i]=pool.ax[i];
                ay[i]=pool.ay[i];
            }
            conf[0]=radius;
            conf[1]=gravity;

            //calculate idx arr
            int *idx_arr0;
            int *idx_arr1;
            int iter_num_0 = ((bodies-1)*bodies)/2;
            idx_arr0=(int*)malloc(sizeof(int)*iter_num_0);
            idx_arr1=(int*)malloc(sizeof(int)*iter_num_0);
            int x_idx_0=0;
            int y_idx_0=1;
            for (int i=0; i<iter_num_0;i++){
                idx_arr0[i]=x_idx_0;
                idx_arr1[i]=y_idx_0;
                y_idx_0++;
                if (y_idx_0 == current_bodies){
                    x_idx_0++;
                    y_idx_0 = x_idx_0+1;
                    if (x_idx_0==current_bodies) x_idx_0=0;
                }
            }

            //transfer all arr into cuda_arr
            int *idx_arr0_cu;
            int *idx_arr1_cu;

            double *m_cu;
            double *x_cu;
            double *y_cu;
            double *vx_cu;
            double *vy_cu;
            double *ax_cu;
            double *ay_cu;

            double *m_delta_cu;
            double *x_delta_cu;
            double *y_delta_cu;
            double *vx_delta_cu;
            double *vy_delta_cu;
            double *ax_delta_cu;
            double *ay_delta_cu;

            double *conf_cu;

            //    hipMalloc((void**)&pool_arr_cu, sizeof(pool));
            hipMalloc((void**)&idx_arr0_cu, sizeof(double)*iter_num_0);
            hipMalloc((void**)&idx_arr1_cu, sizeof(double)*iter_num_0);
            //    hipMemcpy(pool_arr_cu, pool_arr, sizeof(pool), hipMemcpyHostToDevice);
            hipMemcpy(idx_arr0_cu, idx_arr0, sizeof(double)*iter_num_0, hipMemcpyHostToDevice);
            hipMemcpy(idx_arr1_cu, idx_arr1, sizeof(double)*iter_num_0, hipMemcpyHostToDevice);

            hipMalloc((void**)&m_cu,sizeof(double)*bodies);
            hipMalloc((void**)&x_cu,sizeof(double)*bodies);
            hipMalloc((void**)&y_cu,sizeof(double)*bodies);
            hipMalloc((void**)&vx_cu,sizeof(double)*bodies);
            hipMalloc((void**)&vy_cu,sizeof(double)*bodies);
            hipMalloc((void**)&ax_cu,sizeof(double)*bodies);
            hipMalloc((void**)&ay_cu,sizeof(double)*bodies);

            hipMalloc((void**)&m_delta_cu,sizeof(double)*bodies);
            hipMalloc((void**)&x_delta_cu,sizeof(double)*bodies);
            hipMalloc((void**)&y_delta_cu,sizeof(double)*bodies);
            hipMalloc((void**)&vx_delta_cu,sizeof(double)*bodies);
            hipMalloc((void**)&vy_delta_cu,sizeof(double)*bodies);
            hipMalloc((void**)&ax_delta_cu,sizeof(double)*bodies);
            hipMalloc((void**)&ay_delta_cu,sizeof(double)*bodies);

            hipMemcpy(m_cu, m, sizeof(double)*bodies, hipMemcpyHostToDevice);
            hipMemcpy(x_cu, x, sizeof(double)*bodies, hipMemcpyHostToDevice);
            hipMemcpy(y_cu, y, sizeof(double)*bodies, hipMemcpyHostToDevice);
            hipMemcpy(vx_cu, vx, sizeof(double)*bodies, hipMemcpyHostToDevice);
            hipMemcpy(vy_cu, vy, sizeof(double)*bodies, hipMemcpyHostToDevice);
            hipMemcpy(ax_cu, ax, sizeof(double)*bodies, hipMemcpyHostToDevice);
            hipMemcpy(ay_cu, ay, sizeof(double)*bodies, hipMemcpyHostToDevice);

            hipMemcpy(m_delta_cu, m_delta, sizeof(double)*bodies, hipMemcpyHostToDevice);
            hipMemcpy(x_delta_cu, x_delta, sizeof(double)*bodies, hipMemcpyHostToDevice);
            hipMemcpy(y_delta_cu, y_delta, sizeof(double)*bodies, hipMemcpyHostToDevice);
            hipMemcpy(vx_delta_cu, vx_delta, sizeof(double)*bodies, hipMemcpyHostToDevice);
            hipMemcpy(vy_delta_cu, vy_delta, sizeof(double)*bodies, hipMemcpyHostToDevice);
            hipMemcpy(ax_delta_cu, ax_delta, sizeof(double)*bodies, hipMemcpyHostToDevice);
            hipMemcpy(ay_delta_cu, ay_delta, sizeof(double)*bodies, hipMemcpyHostToDevice);

            hipMalloc((void**)&conf_cu,sizeof(double)*2);
            hipMemcpy(conf_cu, conf, sizeof(double)*2, hipMemcpyHostToDevice);

            //kernel calculation
            //dim3 dimGrid(iter_num_0);
            int* start_idx_arr_cu;
            int *start_idx_arr = (int*)malloc(sizeof(int)*2);
            hipMalloc((void**)&start_idx_arr_cu,sizeof(int)*2);

//            int num_threads=3;
            dim3 dimGrid(num_threads);
            dim3 dimBlock(1);

            int small_iter_num = iter_num_0/num_threads;
            if (iter_num_0%num_threads!=0) small_iter_num=small_iter_num+1;

            auto begin = std::chrono::high_resolution_clock::now();
            for (int iter=0; iter<small_iter_num; iter++){
                start_idx_arr[0] = iter*num_threads;
                start_idx_arr[1] = iter_num_0;
                hipMemcpy(start_idx_arr_cu, start_idx_arr, sizeof(int)*2, hipMemcpyHostToDevice);
                //cout<<iter*num_threads<<endl;
                cuda_calaulate<<<dimGrid, dimBlock>>>(start_idx_arr_cu, idx_arr0_cu, idx_arr1_cu, m_cu, x_cu, y_cu, vx_cu, vy_cu, conf_cu, x_delta_cu,y_delta_cu,vx_delta_cu, vy_delta_cu,ax_delta_cu,ay_delta_cu);
            }


            hipMemcpy(m_delta, m_delta_cu, sizeof(double)*bodies, hipMemcpyDeviceToHost);
            hipMemcpy(x_delta, x_delta_cu, sizeof(double)*bodies, hipMemcpyDeviceToHost);
            hipMemcpy(y_delta, y_delta_cu, sizeof(double)*bodies, hipMemcpyDeviceToHost);
            hipMemcpy(vx_delta, vx_delta_cu, sizeof(double)*bodies, hipMemcpyDeviceToHost);
            hipMemcpy(vy_delta, vy_delta_cu, sizeof(double)*bodies, hipMemcpyDeviceToHost);
            hipMemcpy(ax_delta, ax_delta_cu, sizeof(double)*bodies, hipMemcpyDeviceToHost);
            hipMemcpy(ay_delta, ay_delta_cu, sizeof(double)*bodies, hipMemcpyDeviceToHost);

            for (int i=0; i<bodies;i++){
                pool.m[i]+=m_delta[i];
                pool.x[i]+=x_delta[i];
                pool.y[i]+=y_delta[i];
                pool.vx[i]+=vx_delta[i];
                pool.vy[i]+=vy_delta[i];
                pool.ax[i]+=ax_delta[i];
                pool.ay[i]+=ay_delta[i];
            }
            auto end = std::chrono::high_resolution_clock::now();
            size_t duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count();
            std::cout<<"calculation time: "<<duration<<" nanoseconds per iteration."<<std::endl;

            for (size_t i = 0; i < pool.size(); ++i) {
                auto body = pool.get_body(i);
                body.update_for_tick(elapse, space, radius);
            }



            for (size_t i = 0; i < pool.size(); ++i) {
                auto body = pool.get_body(i);
                std::cout<<i<<": "<<" "<<body.get_x()<<" "<<body.get_y()<<" "<<body.get_vx()<<" "<<body.get_vy()<<" "<<body.get_ax()<<" "<<body.get_ay()<<std::endl;
                auto x = p.x + static_cast<float>(body.get_x());
                auto y = p.y + static_cast<float>(body.get_y());
                draw_list->AddCircleFilled(ImVec2(x, y), radius, ImColor{color});
            }
        }
        ImGui::End();
    });
}

